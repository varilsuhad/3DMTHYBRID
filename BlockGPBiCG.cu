#include "hip/hip_runtime.h"
//Created by Deniz Varilsuha
//email: deniz.varilsuha@itu.edu.tr

// To compile the code use the following line in Matlab's command line (change the paths if necessary)
// mexcuda -R2018a BlockGPBiCG.cu -I"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\include" -L"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.3\lib\x64" NVCCFLAGS='"-Wno-deprecated-gpu-targets --gpu-architecture=compute_61  --gpu-code=sm_61,sm_86,sm_89 -use_fast_math -extra-device-vectorization"' -lcusparse -lcublas

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"

void mexFunction(int nleft, mxArray * plhs[], int nright, const mxArray * prhs[])
{

if(nright!=11){
    printf("right hand side is incorrect\n");
    return;
}

mxGPUArray const *row_csr = mxGPUCreateFromMxArray(prhs[0]);
mxGPUArray const *col = mxGPUCreateFromMxArray(prhs[1]);
mxGPUArray const *val = mxGPUCreateFromMxArray(prhs[2]);
mxGPUArray const *b = mxGPUCreateFromMxArray(prhs[3]);
mxGPUArray const *row_csrM = mxGPUCreateFromMxArray(prhs[4]);
mxGPUArray const *colM = mxGPUCreateFromMxArray(prhs[5]);
mxGPUArray const *valM = mxGPUCreateFromMxArray(prhs[6]);
double* tol = mxGetPr(prhs[7]);
double* maxiter = mxGetPr(prhs[8]);
double* stagdetect = mxGetPr(prhs[9]);
mxGPUArray *xi = (mxGPUArray*)mxGPUCreateFromMxArray(prhs[10]);

mwSize N=mxGPUGetNumberOfElements(row_csr)-1;
mwSize ndim = 2;
mwSize dims[2] = {2*N,1};
mwSize nnzM = mxGPUGetNumberOfElements(valM);
mwSize nnz = mxGPUGetNumberOfElements(val);
mwSize dims2[2] = {nnzM,1};

int i,c=0,is;
double res[2000];
double nb,nAx;   
const hipDoubleComplex* d_b = (hipDoubleComplex*)mxGPUGetDataReadOnly(b);   
double resl=1000;

const int* d_row_csrM = (int*)mxGPUGetDataReadOnly(row_csrM);
const int* d_colM = (int*)mxGPUGetDataReadOnly(colM);
const int* d_row_csr = (int*)mxGPUGetDataReadOnly(row_csr);
const int* d_col = (int*)mxGPUGetDataReadOnly(col);

mxGPUArray *y=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *xr=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES); //return x values
mxGPUArray *x0=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *r0=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);    
mxGPUArray *z=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *t=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *wp=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *u=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *p=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *tp=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *Ap=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *AMp=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *tp0=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *At=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *r=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *r0p=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);
mxGPUArray *r0b=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);

hipDoubleComplex* d_r0p = (hipDoubleComplex*)mxGPUGetData(r0p);   
hipDoubleComplex* d_r0b = (hipDoubleComplex*)mxGPUGetData(r0b);
hipDoubleComplex* d_wp = (hipDoubleComplex*)mxGPUGetData(wp);   
hipDoubleComplex* d_u = (hipDoubleComplex*)mxGPUGetData(u);
hipDoubleComplex* d_p = (hipDoubleComplex*)mxGPUGetData(p);   
hipDoubleComplex* d_tp = (hipDoubleComplex*)mxGPUGetData(tp);
hipDoubleComplex* d_Ap = (hipDoubleComplex*)mxGPUGetData(Ap);    
hipDoubleComplex* d_AMp = (hipDoubleComplex*)mxGPUGetData(AMp);
hipDoubleComplex* d_tp0 = (hipDoubleComplex*)mxGPUGetData(tp0);
hipDoubleComplex* d_At = (hipDoubleComplex*)mxGPUGetData(At);
hipDoubleComplex* d_r = (hipDoubleComplex*)mxGPUGetData(r);
hipDoubleComplex* d_r0 = (hipDoubleComplex*)mxGPUGetData(r0);   
hipDoubleComplex* d_y = (hipDoubleComplex*)mxGPUGetData(y);   
hipDoubleComplex* d_valM = (hipDoubleComplex*)mxGPUGetDataReadOnly(valM);
hipDoubleComplex* d_val = (hipDoubleComplex*)mxGPUGetDataReadOnly(val);    
hipDoubleComplex* d_x0 = (hipDoubleComplex*)mxGPUGetData(x0); 
hipDoubleComplex* d_xi = (hipDoubleComplex*)mxGPUGetData(xi);     
hipDoubleComplex* d_xr = (hipDoubleComplex*)mxGPUGetData(xr);        
hipDoubleComplex* d_z = (hipDoubleComplex*)mxGPUGetData(z);   
hipDoubleComplex* d_t = (hipDoubleComplex*)mxGPUGetData(t);

hipDoubleComplex beta = make_hipDoubleComplex(0.0f, 0.0f);
hipDoubleComplex ro0 = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex ro1 = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex ro2 = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex ro3 = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex ro4 = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex ro5 = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex alt = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex ust1 = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex ust2 = make_hipDoubleComplex(1.0f, 0.0f);

hipDoubleComplex alfa = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex minusone = make_hipDoubleComplex(-1.0f, 0.0f);    
hipDoubleComplex w = make_hipDoubleComplex(0.0f, 0.0f);
hipDoubleComplex nu = make_hipDoubleComplex(0.0f, 0.0f);
hipDoubleComplex one = make_hipDoubleComplex(1.0f, 0.0f);
hipDoubleComplex zero = make_hipDoubleComplex(0.0f, 0.0f);
hipDoubleComplex nalfa = make_hipDoubleComplex(1.0f, 0.0f);
   
mxGPUArray *zara=mxGPUCreateGPUArray(ndim,dims,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);     
mxGPUArray *valM2s=mxGPUCreateGPUArray(ndim,dims2,mxDOUBLE_CLASS,mxCOMPLEX,MX_GPU_INITIALIZE_VALUES);    
hipDoubleComplex* d_zara = (hipDoubleComplex*)mxGPUGetData(zara);
hipDoubleComplex* d_valM2s = (hipDoubleComplex*)mxGPUGetData(valM2s);        

// CUBLAS APIs
hipblasHandle_t cublashandle=NULL;
hipblasStatus_t status;
status = hipblasCreate(&cublashandle);
if (status != HIPBLAS_STATUS_SUCCESS) {printf("!!!! CUBLAS initialization error\n");return;}

// CUSPARSE APIs
hipsparseHandle_t cusparsehandle = NULL;
hipsparseStatus_t status2;
status2=hipsparseCreate(&cusparsehandle);
if (status2 != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse initialization error\n");return;}
hipsparseMatDescr_t descr_M = 0;
csrilu02Info_t info_M  = 0;
int pBufferSize_M;
int pBufferSize_A;
size_t bufferSizeL,bufferSizeU;    
void *pBuffer = 0,*d_bufferLU,*d_bufferLs,*d_bufferUs;
// descriptor'lar
status2=hipsparseCreateMatDescr(&descr_M);
status2=hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
status2=hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);
status2=hipsparseCreateCsrilu02Info(&info_M);
   
hipsparseFillMode_t fill_lower    = HIPSPARSE_FILL_MODE_LOWER;
hipsparseDiagType_t diag_unit     = HIPSPARSE_DIAG_TYPE_UNIT;
hipsparseFillMode_t fill_upper    = HIPSPARSE_FILL_MODE_UPPER;
hipsparseDiagType_t diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

hipsparseSpSMDescr_t spsvDescrLs, spsvDescrUs;
    
hipsparseSpMatDescr_t matA;
hipsparseDnMatDescr_t matp;
hipsparseDnMatDescr_t mattp, matAt;
hipsparseDnMatDescr_t matx0, matr0,matr0p,matAMp;
hipsparseSpMatDescr_t matM_lowers, matM_uppers;
hipsparseDnMatDescr_t matAp,matzara;
    
status2=hipsparseCreateCsr(&matA, N, N, nnz, (void*)d_row_csr, (void*)d_col, d_val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F) ;
status2=hipsparseCreateDnMat(&matp, N,2,N, d_p, HIP_C_64F,HIPSPARSE_ORDER_COL);
status2=hipsparseCreateDnMat(&matAp, N,2,N, d_Ap, HIP_C_64F,HIPSPARSE_ORDER_COL);
status2=hipsparseCreateDnMat(&mattp, N,2,N, d_tp, HIP_C_64F,HIPSPARSE_ORDER_COL);
status2=hipsparseCreateDnMat(&matAt, N,2,N, d_At, HIP_C_64F,HIPSPARSE_ORDER_COL);
status2=hipsparseCreateDnMat(&matx0, N,2,N, d_x0, HIP_C_64F,HIPSPARSE_ORDER_COL);
status2=hipsparseCreateDnMat(&matr0, N,2,N, d_r0, HIP_C_64F,HIPSPARSE_ORDER_COL);
status2=hipsparseCreateDnMat(&matr0p, N,2,N, d_r0p, HIP_C_64F,HIPSPARSE_ORDER_COL);
status2=hipsparseCreateDnMat(&matAMp, N,2,N, d_AMp, HIP_C_64F,HIPSPARSE_ORDER_COL);
status2=hipsparseCreateDnMat(&matzara, N,2,N, d_zara, HIP_C_64F,HIPSPARSE_ORDER_COL);

hipsparseSpMMAlg_t alg=HIPSPARSE_SPMM_CSR_ALG1;  
hipsparseSpSMAlg_t alg2=HIPSPARSE_SPSM_ALG_DEFAULT;

status2=hipsparseSpMM_bufferSize(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, matp, &zero, matAp, HIP_C_64F, alg , (size_t*)&pBufferSize_A);
if (status2 != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse Ax buffer error\n");return;}
hipMalloc((void**)&pBuffer, pBufferSize_A);

status2=hipsparseSpMM_preprocess(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, mattp, &zero, matAt, HIP_C_64F, alg , pBuffer);

hipsparseMatDescr_t matLU;    
hipsparseCreateMatDescr(&matLU);
hipsparseSetMatType(matLU, HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatIndexBase(matLU, HIPSPARSE_INDEX_BASE_ZERO);

status=hipblasZcopy(cublashandle, nnzM, d_valM, 1, d_valM2s, 1);

status2=hipsparseZcsrilu02_bufferSize(cusparsehandle, N, nnzM, matLU, d_valM2s, d_row_csrM, d_colM, info_M, &pBufferSize_M);
if (status2 != HIPSPARSE_STATUS_SUCCESS) { printf("cusparse ilu error\n");return;}
hipMalloc((void**)&d_bufferLU, pBufferSize_M);

hipsparseZcsrilu02_analysis( cusparsehandle, N, nnzM, descr_M, d_valM2s, d_row_csrM, d_colM, info_M, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferLU);
hipsparseZcsrilu02( cusparsehandle, N, nnzM, matLU, d_valM2s, d_row_csrM, d_colM, info_M, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, d_bufferLU);

hipsparseSpSM_createDescr(&spsvDescrLs);
hipsparseSpSM_createDescr(&spsvDescrUs);    

///////////////////////////////

hipsparseCreateCsr(&matM_lowers, N, N, nnzM, (void*)d_row_csrM, (void*)d_colM, d_valM2s, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);
hipsparseSpMatSetAttribute(matM_lowers,HIPSPARSE_SPMAT_FILL_MODE, &fill_lower, sizeof(fill_lower));
hipsparseSpMatSetAttribute(matM_lowers, HIPSPARSE_SPMAT_DIAG_TYPE, &diag_unit, sizeof(diag_unit));

hipsparseCreateCsr(&matM_uppers, N, N, nnzM, (void*)d_row_csrM, (void*)d_colM, d_valM2s, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);
hipsparseSpMatSetAttribute(matM_uppers, HIPSPARSE_SPMAT_FILL_MODE, &fill_upper, sizeof(fill_upper));
hipsparseSpMatSetAttribute(matM_uppers, HIPSPARSE_SPMAT_DIAG_TYPE, &diag_non_unit,  sizeof(diag_non_unit)); 

hipsparseSpSM_bufferSize( cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_lowers, matr0, matzara, HIP_C_64F, alg2, spsvDescrLs, &bufferSizeL);
hipMalloc(&d_bufferLs, bufferSizeL);
hipsparseSpSM_bufferSize(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_uppers, matr0, matzara, HIP_C_64F, alg2, spsvDescrUs, &bufferSizeU);
hipMalloc(&d_bufferUs, bufferSizeU);   

hipsparseSpSM_analysis(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_lowers, matr0, matzara, HIP_C_64F, alg2, spsvDescrLs, d_bufferLs);
hipsparseSpSM_analysis(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_uppers, matr0, matzara, HIP_C_64F, alg2, spsvDescrUs, d_bufferUs);
    
/////////////////////////////////

// xi ->x0 kopyala
status=hipblasZcopy(cublashandle, 2*N, d_xi, 1, d_x0, 1);

// nb=norm(b);
status=hipblasDznrm2(cublashandle,2*N, d_b, 1, &nb);

// r0=(b-A*x0);
status=hipblasZcopy(cublashandle, 2*N, d_b, 1, d_r0, 1);
status2=hipsparseSpMM(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusone, matA, matx0, &one, matr0, HIP_C_64F, alg , pBuffer);    

hipsparseSpSM_solve(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_lowers, matr0, matzara, HIP_C_64F, alg2, spsvDescrLs);
hipsparseSpSM_solve(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_uppers, matzara, matr0p, HIP_C_64F, alg2, spsvDescrUs);

//r0b=M2\(M1\r0p); //modladım //r0b=r0p
status=hipblasZcopy(cublashandle, 2*N, d_r0p, 1, d_r0b, 1);

// FOR LOOP
for (i=0;i<(int)(*maxiter);++i)
{
//p=r0p+beta*(p-u);
status=hipblasZaxpy(cublashandle, 2*N, &minusone, d_u, 1, d_p, 1);
status=hipblasZscal(cublashandle, 2*N, &beta, d_p, 1);
status=hipblasZaxpy(cublashandle, 2*N, &one, d_r0p, 1, d_p, 1);
//Ap=A*p;
status2=hipsparseSpMM(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, matp, &zero, matAp, HIP_C_64F, alg , pBuffer);
  
hipsparseSpSM_solve(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_lowers, matAp, matzara, HIP_C_64F, alg2, spsvDescrLs);
hipsparseSpSM_solve(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_uppers, matzara, matAMp, HIP_C_64F, alg2, spsvDescrUs);

// alfa=(r0b'*r0)/(r0b'*Ap);
status=hipblasZdotc(cublashandle,2*N, d_r0b, 1, d_r0, 1, &ro0);    
status=hipblasZdotc(cublashandle,2*N, d_r0b, 1, d_Ap, 1, &ro1);    
alfa=hipCdiv(ro0,ro1);

//y=t-r0+alfa*(Ap-wp);
status=hipblasZcopy(cublashandle, 2*N, d_Ap, 1, d_y, 1);
status=hipblasZaxpy(cublashandle, 2*N, &minusone, d_wp, 1, d_y, 1);
status=hipblasZscal(cublashandle, 2*N, &alfa, d_y, 1);
status=hipblasZaxpy(cublashandle, 2*N, &minusone, d_r0, 1, d_y, 1);
status=hipblasZaxpy(cublashandle, 2*N, &one, d_t, 1, d_y, 1);

//t=r0-alfa*Ap;
status=hipblasZcopy(cublashandle, 2*N, d_r0, 1, d_t, 1);
nalfa=hipCmul(minusone,alfa);
status=hipblasZaxpy(cublashandle, 2*N, &nalfa, d_Ap, 1, d_t, 1);

//tp0=tp;
status=hipblasZcopy(cublashandle, 2*N, d_tp, 1, d_tp0, 1);

//tp=r0p-alfa*AMp;
status=hipblasZcopy(cublashandle, 2*N, d_r0p, 1, d_tp, 1);
status=hipblasZaxpy(cublashandle, 2*N, &nalfa, d_AMp, 1, d_tp, 1);
    
//At=A*tp;
status2=hipsparseSpMM(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, mattp, &zero, matAt, HIP_C_64F, alg , pBuffer);
// if (status2 != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse Ax carpim  error\n");return;}

if(i==0){
//w=(At'*t)/(At'*At);   n=0;    
status=hipblasZdotc(cublashandle,2*N, d_At, 1, d_t, 1, &ro0);    
status=hipblasZdotc(cublashandle,2*N, d_At, 1, d_At, 1, &ro1);   
w=hipCdiv(ro0,ro1);
nu = make_hipDoubleComplex(0.0f, 0.0f);

}else{
//w=((y'*y)*(At'*t)-(y'*t)*(At'*y))/((At'*At)*(y'*y)-(y'*At)*(At'*y));
//n=((At'*At)*(y'*t)-(y'*At)*(At'*t))/((At'*At)*(y'*y)-(y'*At)*(At'*y));    
status=hipblasZdotc(cublashandle,2*N, d_y, 1, d_y, 1, &ro0);    
status=hipblasZdotc(cublashandle,2*N, d_At, 1, d_t, 1, &ro1);    
status=hipblasZdotc(cublashandle,2*N, d_y, 1, d_t, 1, &ro2);    
status=hipblasZdotc(cublashandle,2*N, d_At, 1, d_y, 1, &ro3);    
status=hipblasZdotc(cublashandle,2*N, d_At, 1, d_At, 1, &ro4);    
status=hipblasZdotc(cublashandle,2*N, d_y, 1, d_At, 1, &ro5);    

alt=hipCsub(hipCmul(ro4,ro0),hipCmul(ro5,ro3));
ust1=hipCsub(hipCmul(ro0,ro1),hipCmul(ro2,ro3));
ust2=hipCsub(hipCmul(ro4,ro2),hipCmul(ro5,ro1));

w=hipCdiv(ust1,alt);  
nu=hipCdiv(ust2,alt);    
}

//u=w*AMp+n*(tp0-r0p+beta*u);
status=hipblasZscal(cublashandle, 2*N, &beta, d_u, 1);
status=hipblasZaxpy(cublashandle, 2*N, &minusone, d_r0p, 1, d_u, 1);
status=hipblasZaxpy(cublashandle, 2*N, &one, d_tp0, 1, d_u, 1);
status=hipblasZscal(cublashandle, 2*N, &nu, d_u, 1);
status=hipblasZaxpy(cublashandle, 2*N, &w, d_AMp, 1, d_u, 1);

//z=w*r0p+n*z-alfa*u;
status=hipblasZscal(cublashandle, 2*N, &nu, d_z, 1);
status=hipblasZaxpy(cublashandle, 2*N, &w, d_r0p, 1, d_z, 1);
status=hipblasZaxpy(cublashandle, 2*N, &nalfa, d_u, 1, d_z, 1);

//x0=x0+alfa*p+z;
status=hipblasZaxpy(cublashandle, 2*N, &alfa, d_p, 1, d_x0, 1);
status=hipblasZaxpy(cublashandle, 2*N, &one, d_z, 1, d_x0, 1);

//r=r0;
status=hipblasZcopy(cublashandle, 2*N, d_r0, 1, d_r, 1);

//r0=t-n*y-w*At;
status=hipblasZcopy(cublashandle, 2*N, d_t, 1, d_r0, 1);
ro0=hipCmul(minusone,nu);
ro1=hipCmul(minusone,w);
status=hipblasZaxpy(cublashandle, 2*N, &ro0, d_y, 1, d_r0, 1);
status=hipblasZaxpy(cublashandle, 2*N, &ro1, d_At, 1, d_r0, 1);

// //r0p=M2\(M1\r0);
hipsparseSpSM_solve(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_lowers, matr0, matzara, HIP_C_64F, alg2, spsvDescrLs);
hipsparseSpSM_solve(cusparsehandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matM_uppers, matzara, matr0p, HIP_C_64F, alg2, spsvDescrUs);

//beta=alfa/w*(r0b'*r0)/(r0b'*r);
status=hipblasZdotc(cublashandle, 2*N, d_r0b, 1, d_r0, 1, &ro0);    
status=hipblasZdotc(cublashandle, 2*N, d_r0b, 1, d_r, 1, &ro1);    
ro2=hipCdiv(alfa,w);
ro3=hipCdiv(ro0,ro1);
beta=hipCmul(ro2,ro3);    

//wp=At+beta*Ap;
status=hipblasZcopy(cublashandle, 2*N, d_At, 1, d_wp, 1);
status=hipblasZaxpy(cublashandle, 2*N, &beta, d_Ap, 1, d_wp, 1);

// relres=norm(r0)/nb;
status=hipblasDznrm2(cublashandle, 2*N, d_r0, 1, &nAx);
res[i]=nAx/nb;


if(res[i]<=*tol){
    status=hipblasZcopy(cublashandle, 2*N, d_x0, 1, d_xr, 1);
    is=i;
    break;
}

// x0->xr    
if(resl>res[i]){
status=hipblasZcopy(cublashandle, 2*N, d_x0, 1, d_xr, 1);
resl=res[i];
c=0;
is=i;    
}else{
c=c+1;
  if(c>(int)(*stagdetect)){
      break;
    }
   
}
}

plhs[0] = mxGPUCreateMxArrayOnGPU(xr);

int nn;
if(i==(int)(*maxiter)){    
nn=i;
}else{
nn=i+1;
}

plhs[1] = mxCreateDoubleMatrix(nn, 1, mxREAL);
double* py = mxGetPr(plhs[1]);
for (i=0;i<nn;++i){
py[i]=res[i];
}

plhs[2] = mxCreateDoubleMatrix(1, 1, mxREAL);
double* py2 = mxGetPr(plhs[2]);
py2[0]=res[is];

hipFree(d_bufferLs);
hipFree(d_bufferUs);
hipFree(d_bufferLU);

hipsparseSpSM_destroyDescr(spsvDescrUs);   
hipsparseSpSM_destroyDescr(spsvDescrLs);    
hipsparseDestroyMatDescr(matLU);

mxGPUDestroyGPUArray(xi);    
mxGPUDestroyGPUArray(row_csr);
mxGPUDestroyGPUArray(col);
mxGPUDestroyGPUArray(val);
mxGPUDestroyGPUArray(b);
mxGPUDestroyGPUArray(row_csrM);
mxGPUDestroyGPUArray(colM);
mxGPUDestroyGPUArray(valM);

mxGPUDestroyGPUArray(zara);
mxGPUDestroyGPUArray(valM2s); 
mxGPUDestroyGPUArray(y);
mxGPUDestroyGPUArray(xr);    
mxGPUDestroyGPUArray(x0);
mxGPUDestroyGPUArray(r0);
mxGPUDestroyGPUArray(z);
mxGPUDestroyGPUArray(t);
mxGPUDestroyGPUArray(wp);
mxGPUDestroyGPUArray(u);
mxGPUDestroyGPUArray(p);
mxGPUDestroyGPUArray(tp);
mxGPUDestroyGPUArray(Ap);
mxGPUDestroyGPUArray(AMp);
mxGPUDestroyGPUArray(tp0);
mxGPUDestroyGPUArray(At);
mxGPUDestroyGPUArray(r);
mxGPUDestroyGPUArray(r0p);
mxGPUDestroyGPUArray(r0b);

hipsparseDestroySpMat(matA);
hipsparseDestroySpMat(matM_lowers);
hipsparseDestroySpMat(matM_uppers);    
hipsparseDestroyDnMat(matp);    
hipsparseDestroyDnMat(matAp);    
hipsparseDestroyDnMat(mattp);    
hipsparseDestroyDnMat(matAt);    
hipsparseDestroyDnMat(matx0);    
hipsparseDestroyDnMat(matr0);    
hipsparseDestroyDnMat(matr0p);    
hipsparseDestroyDnMat(matAMp);    
hipsparseDestroyDnMat(matzara);    
    
hipsparseDestroyMatDescr(descr_M);
hipsparseDestroyCsrilu02Info(info_M);

status=hipblasDestroy(cublashandle);
if (status != HIPBLAS_STATUS_SUCCESS) {printf("!!!! hipblasDestroy error\n");return;}
status2=hipsparseDestroy(cusparsehandle);
if (status2 != HIPSPARSE_STATUS_SUCCESS) {printf("!!!! cusparse destroy error\n");return;}
hipFree(pBuffer);

hipDeviceSynchronize();
return;
}


